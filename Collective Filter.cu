

// the initial letter of GPU function is in UPPER CASE
// the initial letter of CPU function is in LOWER CASE

// Global variable can not be used in kernel
#include <stdio.h>
#include <hip/hip_runtime.h>

#define Max 20000
// 80000 = 64 * 1250
#define testMax 2000
// 20000 ~= 64 * 320
#define Maxuser 2000

// search for the first position of every userId
__global__ void  Search(int* userId, int* userPos){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j;
	if(i < 20000){
		j = userId[i];
		if(i > 0){
			if(userId[i] != userId[i-1]){
				userPos[j] = i;
			}
        }
	}
	__syncthreads();
}
// dont't get the sum in parallel right now
__global__ void Get_ravg(double* d_Ravg, int userNum,double* d_rating, int* d_userPos){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j;

	if(i < userNum+1){
		for(j = d_userPos[i]; j < d_userPos[i+1]; j++){
			d_Ravg[i] += d_rating[j];
		}
		d_Ravg[i] = d_Ravg[i]/(d_userPos[i+1] - d_userPos[i]);
	}
	__syncthreads();
}

__global__ void Get_ratingM(double* d_Ravg, int* d_userId, double* d_rating, double* d_ratingM, int k){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = 10000;
    if(i < j){
        d_ratingM[i] = d_rating[i] - d_Ravg[d_userId[i]];
//        d_ratingM[i] = 6.66;
    }
    __syncthreads();
}

int* get_userPos(int userNum,int* h_userPos, int* h_userId);
void read_training_set(int* h_userId, int* h_movieId, double* h_rating, int timestamp);
void read_test_set(int* h_testuserId, int* h_testmovieId, double* h_testrating, int timestamp);
double* get_ravg(int userNum, double* h_rating, int* d_userPos, double* h_Ravg);
double* get_ratingM(double* d_Ravg, int* h_userId, double* h_rating, double* h_ratingM);

int main(){
    // Read data from training set
	int h_userId[Max], h_movieId[Max] ,timestamp;
    double h_rating[Max];
    read_training_set(h_userId, h_movieId, h_rating, timestamp);

    // Read data from test set
    int h_testuserId[testMax], h_testmovieId[testMax];
    double h_testrating[testMax];
    read_test_set(h_testuserId, h_testmovieId, h_testrating, timestamp);


    // userPos is an array, storing the beginning position of the user.
    // Ex: userPos[2] = 9 means userId 2 starts from 9
    int userNum = h_userId[Max-1];
    int h_userPos[Maxuser] = {0};
    int *d_userPos;
    d_userPos = get_userPos(userNum, h_userPos, h_userId);
    for(int i = 1; i < userNum; i++){
        printf("%d\n userpos",h_userPos[i]);
//        printf("%d\n",h_userId[h_userPos[i]]);
//        printf("%d: %lf\n",h_userPos[i], h_Ravg[i]);
    }

    // get R average of every user
    double h_Ravg[Maxuser] = {0.0};
    double* d_Ravg;
    d_Ravg = get_ravg(userNum, h_rating, d_userPos, h_Ravg);


    // ratingm means that Rui - Ravg
    // After we get ratingm, we could figure out the Sim of i and j by scalar product (内积）
    // with sparse matrix library “cusparse”
    double h_ratingM[Max];
    double* d_ratingM;
    d_ratingM = get_ratingM(d_Ravg, h_userId, h_rating, h_ratingM);



    //just a test

    printf("userNum: %d\n",userNum);


     for(int i = 1; i < userNum; i++){
//         printf("%d:",h_userPos[i]);
//         printf("%d\n",h_userId[h_userPos[i]]);
         printf("userId: %d %d: %lf\n",h_userId[h_userPos[i]],h_userPos[i], h_Ravg[i]);
         printf("h_rating: %lf, d_ratingM: %lf\n", h_rating[i], h_ratingM[i]);
     }

    return 0;
}



// leave the ratingM in the global GPU memory
double* get_ratingM(double* d_Ravg, int* h_userId, double* h_rating, double* h_ratingM){
	int k;
	k = Max;

    int* d_userId;
    double* d_rating;
    double* d_ratingM;

    hipMalloc((void **)&d_userId, Max);
    hipMalloc((void **)&d_rating, Max);
    hipMalloc((void **)&d_ratingM, Max);

    hipMemcpy(d_rating, h_rating, Max, hipMemcpyHostToDevice);
	hipMemcpy(d_userId, h_userId, Max, hipMemcpyHostToDevice);
    Get_ratingM<<<Max/128 + 1,128>>>(d_Ravg, d_userId, d_rating, d_ratingM, k);

    hipMemcpy(h_ratingM, d_ratingM, Max, hipMemcpyDeviceToHost);

    hipFree(d_userId);
    hipFree(d_rating);

    return d_ratingM;
}

// get R average of every user
double* get_ravg(int userNum, double* h_rating, int* d_userPos, double* h_Ravg){

    double* d_Ravg;
    double* d_rating;

    hipMalloc((void **)&d_Ravg, userNum);
    hipMalloc((void **)&d_rating, Max);

    hipMemcpy(d_rating, h_rating, Max, hipMemcpyHostToDevice);
	hipMemcpy(d_Ravg, h_Ravg, Max, hipMemcpyHostToDevice);
    Get_ravg<<<10,128>>>(d_Ravg, userNum, d_rating, d_userPos);

    hipMemcpy(h_Ravg, d_Ravg, userNum, hipMemcpyDeviceToHost);
    hipFree(d_rating);
    return d_Ravg;
}

// get userPos by GPU computation
int* get_userPos(int userNum,int* h_userPos, int* h_userId){
    // initialize
	h_userPos[userNum+1] = Max;
//    h_userPos[0] = -1;  // -1: does not exist

    // declare device variable and allocate device memroy
    int* d_userPos;
    int* d_userId;
    hipMalloc((void **)&d_userPos, Maxuser);
    hipMalloc((void **)&d_userId, Max);

    hipMemcpy(d_userPos, h_userPos, Maxuser, hipMemcpyHostToDevice);
    hipMemcpy(d_userId, h_userId, Max, hipMemcpyHostToDevice);

    Search<<<Max/128+1,128>>>(d_userId, d_userPos);

    hipMemcpy(h_userPos, d_userPos, Maxuser, hipMemcpyDeviceToHost);
    hipFree(d_userId);
    return d_userPos;
}

// Read data from training set
void read_training_set(int* h_userId, int* h_movieId, double* h_rating, int timestamp){
    FILE *p = NULL;
    p = fopen("/usr/local/MovieLens/ml-100k/u1.base","r");
    if(p == NULL){
        printf("Read Error!\n");
        return;
    }
	for(int i = 0; i < Max; i++){
        fscanf(p,"%d %d %lf %d",&h_userId[i] , &h_movieId[i], &h_rating[i], &timestamp);
		// fscanf(p,"%d%*2c%d%*2c%lf%*2c%d",&h_userId[i] , &h_movieId[i], &h_rating[i], &timestamp);
//		printf("%d %d %lf\n",h_userId[i] , h_movieId[i], h_rating[i]);
	}
    fclose(p);
    return;
}

// Read data from test set
void read_test_set(int* h_testuserId, int* h_testmovieId, double* h_testrating, int timestamp){
    FILE *p = NULL;
    p = fopen("/usr/local/MovieLens/ml-100k/u1.test","r");
    if(p == NULL){
        printf("Read Error!\n");
        return;
    }
	for(int i = 0; i < testMax; i++){
        fscanf(p,"%d %d %lf %d",&h_testuserId[i] , &h_testmovieId[i], &h_testrating[i], &timestamp);
		// fscanf(p,"%d%*2c%d%*2c%lf%*2c%d",&h_userId[i] , &h_movieId[i], &h_rating[i], &timestamp);
		// printf("%d %d %lf\n",h_userId[i] , h_movieId[i], h_rating[i]);
	}
    fclose(p);
    return;
}

// get the MAE

//__global__ MAEcompute(double* pred, double* d_testrating, double* MAE){
//   int i = blockIdx.x * blockDim.x + threadIdx.x;
//   double til;
//
//   if(pred[i] >= test[i]){
//       MAE[i] = pred[i] - d_testrating[i];
//   }else{
//       MAE[i] = d_testrating[i] - pred[i];
//   }
//   __syncthreads();
//}

//
//__device__ sumFunc(double number, int total){
//    i = blockIdx.x * blockDim.x + threadIdx.x;
//    int middle = total/2;
//    if()
//}


